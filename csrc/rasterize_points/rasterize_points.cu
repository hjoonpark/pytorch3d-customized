#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <tuple>
#include "rasterize_points/bitmask.cuh"
#include "rasterize_points/rasterization_utils.cuh"

namespace {
// A little structure for holding details about a pixel.
struct Pix {
  float z; // Depth of the reference point.
  int32_t idx; // Index of the reference point.
  float dist2; // Euclidean distance square to the reference point.
};

__device__ inline bool operator<(const Pix& a, const Pix& b) {
  return a.z < b.z;
}

// This function checks if a pixel given by xy location pxy lies within the
// point with index p and batch index n. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the points which intersect
// with this pixel sorted by closest z distance. If the pixel pxy lies in the
// point, the list (q) is updated and re-orderered in place. In addition
// the auxillary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizePointsNaiveCudaKernel and
// RasterizePointsFineCudaKernel.
template <typename PointQ>
__device__ void CheckPixelInsidePoint(
    const float* points, // (P, 3)
    const int p_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    PointQ& q,
    const float radius2,
    const float xf,
    const float yf,
    const int K) {
  const float px = points[p_idx * 3 + 0];
  const float py = points[p_idx * 3 + 1];
  const float pz = points[p_idx * 3 + 2];
  if (pz < 0)
    return; // Don't render points behind the camera
  const float dx = xf - px;
  const float dy = yf - py;
  const float dist2 = dx * dx + dy * dy;
  if (dist2 < radius2) {
    if (q_size < K) {
      // Just insert it
      q[q_size] = {pz, p_idx, dist2};
      if (pz > q_max_z) {
        q_max_z = pz;
        q_max_idx = q_size;
      }
      q_size++;
    } else if (pz < q_max_z) {
      // Overwrite the old max, and find the new max
      q[q_max_idx] = {pz, p_idx, dist2};
      q_max_z = pz;
      for (int i = 0; i < K; i++) {
        if (q[i].z > q_max_z) {
          q_max_z = q[i].z;
          q_max_idx = i;
        }
      }
    }
  }
}
} // namespace
// ****************************************************************************
// *                          NAIVE RASTERIZATION                             *
// ****************************************************************************

__global__ void RasterizePointsNaiveCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float radius,
    const int N,
    const int S,
    const int K,
    int32_t* point_idxs, // (N, S, S, K)
    float* zbuf, // (N, S, S, K)
    float* pix_dists) { // (N, S, S, K)
  // Simple version: One thread per output pixel
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const float radius2 = radius * radius;
  for (int i = tid; i < N * S * S; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (S * S); // Batch index
    const int pix_idx = i % (S * S);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yi = S - 1 - pix_idx / S;
    const int xi = S - 1 - pix_idx % S;

    const float xf = PixToNdc(xi, S);
    const float yf = PixToNdc(yi, S);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    // TODO(jcjohns) Abstract this out into a standalone data structure
    Pix q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t point_start_idx = cloud_to_packed_first_idx[n];
    const int64_t point_stop_idx = point_start_idx + num_points_per_cloud[n];

    for (int p_idx = point_start_idx; p_idx < point_stop_idx; ++p_idx) {
      CheckPixelInsidePoint(
          points, p_idx, q_size, q_max_z, q_max_idx, q, radius2, xf, yf, K);
    }
    BubbleSort(q, q_size);
    int idx = n * S * S * K + pix_idx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist2;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
RasterizePointsNaiveCuda(
    const torch::Tensor& points, // (P. 3)
    const torch::Tensor& cloud_to_packed_first_idx, // (N)
    const torch::Tensor& num_points_per_cloud, // (N)
    const int image_size,
    const float radius,
    const int points_per_pixel) {
  if (points.ndimension() != 2 || points.size(1) != 3) {
    AT_ERROR("points must have dimensions (num_points, 3)");
  }
  if (num_points_per_cloud.size(0) != cloud_to_packed_first_idx.size(0)) {
    AT_ERROR(
        "num_points_per_cloud must have same size first dimension as cloud_to_packed_first_idx");
  }

  const int N = num_points_per_cloud.size(0); // batch size.
  const int S = image_size;
  const int K = points_per_pixel;

  if (K > kMaxPointsPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxPointsPerPixel;
    AT_ERROR(ss.str());
  }

  auto int_opts = points.options().dtype(torch::kInt32);
  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor point_idxs = torch::full({N, S, S, K}, -1, int_opts);
  torch::Tensor zbuf = torch::full({N, S, S, K}, -1, float_opts);
  torch::Tensor pix_dists = torch::full({N, S, S, K}, -1, float_opts);

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizePointsNaiveCudaKernel<<<blocks, threads>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius,
      N,
      S,
      K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());
  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                          COARSE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizePointsCoarseCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float radius,
    const int N,
    const int P,
    const int S,
    const int bin_size,
    const int chunk_size,
    const int max_points_per_bin,
    int* points_per_bin,
    int* bin_points) {
  extern __shared__ char sbuf[];
  const int M = max_points_per_bin;
  const int num_bins = 1 + (S - 1) / bin_size; // Integer divide round up
  const float half_pix = 1.0f / S; // Size of half a pixel in NDC units

  // This is a boolean array of shape (num_bins, num_bins, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins, num_bins, chunk_size);

  // Have each block handle a chunk of points and build a 3D bitmask in
  // shared memory to mark which points hit which bins.  In this first phase,
  // each thread processes one point at a time. After processing the chunk,
  // one thread is assigned per bin, and the thread counts and writes the
  // points for the bin out to global memory.
  const int chunks_per_batch = 1 + (P - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;
  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch;
    const int chunk_idx = chunk % chunks_per_batch;
    const int point_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t cloud_point_start_idx = cloud_to_packed_first_idx[batch_idx];
    const int64_t cloud_point_stop_idx =
        cloud_point_start_idx + num_points_per_cloud[batch_idx];

    // Have each thread handle a different point within the chunk
    for (int p = threadIdx.x; p < chunk_size; p += blockDim.x) {
      const int p_idx = point_start_idx + p;

      // Check if point index corresponds to the cloud in the batch given by
      // batch_idx.
      if (p_idx >= cloud_point_stop_idx || p_idx < cloud_point_start_idx) {
        continue;
      }

      const float px = points[p_idx * 3 + 0];
      const float py = points[p_idx * 3 + 1];
      const float pz = points[p_idx * 3 + 2];
      if (pz < 0)
        continue; // Don't render points behind the camera.
      const float px0 = px - radius;
      const float px1 = px + radius;
      const float py0 = py - radius;
      const float py1 = py + radius;

      // Brute-force search over all bins; TODO something smarter?
      // For example we could compute the exact bin where the point falls,
      // then check neighboring bins. This way we wouldn't have to check
      // all bins (however then we might have more warp divergence?)
      for (int by = 0; by < num_bins; ++by) {
        // Get y extent for the bin. PixToNdc gives us the location of
        // the center of each pixel, so we need to add/subtract a half
        // pixel to get the true extent of the bin.
        const float by0 = PixToNdc(by * bin_size, S) - half_pix;
        const float by1 = PixToNdc((by + 1) * bin_size - 1, S) + half_pix;
        const bool y_overlap = (py0 <= by1) && (by0 <= py1);

        if (!y_overlap) {
          continue;
        }
        for (int bx = 0; bx < num_bins; ++bx) {
          // Get x extent for the bin; again we need to adjust the
          // output of PixToNdc by half a pixel.
          const float bx0 = PixToNdc(bx * bin_size, S) - half_pix;
          const float bx1 = PixToNdc((bx + 1) * bin_size - 1, S) + half_pix;
          const bool x_overlap = (px0 <= bx1) && (bx0 <= px1);

          if (x_overlap) {
            binmask.set(by, bx, p);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every point in the current chunk. We need to
    // count the number of points in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins * num_bins; byx += blockDim.x) {
      const int by = byx / num_bins;
      const int bx = byx % num_bins;
      const int count = binmask.count(by, bx);
      const int points_per_bin_idx =
          batch_idx * num_bins * num_bins + by * num_bins + bx;

      // This atomically increments the (global) number of points found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_points array for the
      // points in the current chunk that fall into this bin.
      const int start = atomicAdd(points_per_bin + points_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_points.
      int next_idx = batch_idx * num_bins * num_bins * M + by * num_bins * M +
          bx * M + start;
      for (int p = 0; p < chunk_size; ++p) {
        if (binmask.get(by, bx, p)) {
          // TODO: Throw an error if next_idx >= M -- this means that
          // we got more than max_points_per_bin in this bin
          // TODO: check if atomicAdd is needed in line 265.
          bin_points[next_idx] = point_start_idx + p;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

torch::Tensor RasterizePointsCoarseCuda(
    const torch::Tensor& points, // (P, 3)
    const torch::Tensor& cloud_to_packed_first_idx, // (N)
    const torch::Tensor& num_points_per_cloud, // (N)
    const int image_size,
    const float radius,
    const int bin_size,
    const int max_points_per_bin) {
  const int P = points.size(0);
  const int N = num_points_per_cloud.size(0);
  const int num_bins = 1 + (image_size - 1) / bin_size; // divide round up
  const int M = max_points_per_bin;

  if (points.ndimension() != 2 || points.size(1) != 3) {
    AT_ERROR("points must have dimensions (num_points, 3)");
  }
  if (num_bins >= 22) {
    // Make sure we do not use too much shared memory.
    std::stringstream ss;
    ss << "Got " << num_bins << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = points.options().dtype(torch::kInt32);
  torch::Tensor points_per_bin = torch::zeros({N, num_bins, num_bins}, opts);
  torch::Tensor bin_points = torch::full({N, num_bins, num_bins, M}, -1, opts);
  const int chunk_size = 512;
  const size_t shared_size = num_bins * num_bins * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizePointsCoarseCudaKernel<<<blocks, threads, shared_size>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius,
      N,
      P,
      image_size,
      bin_size,
      chunk_size,
      M,
      points_per_bin.contiguous().data_ptr<int32_t>(),
      bin_points.contiguous().data_ptr<int32_t>());
  return bin_points;
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizePointsFineCudaKernel(
    const float* points, // (P, 3)
    const int32_t* bin_points, // (N, B, B, T)
    const float radius,
    const int bin_size,
    const int N,
    const int B, // num_bins
    const int M,
    const int S,
    const int K,
    int32_t* point_idxs, // (N, S, S, K)
    float* zbuf, // (N, S, S, K)
    float* pix_dists) { // (N, S, S, K)
  // This can be more than S^2 if S is not dividable by bin_size.
  const int num_pixels = N * B * B * bin_size * bin_size;
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const float radius2 = radius * radius;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from points and bin_points.
    int i = pid;
    const int n = i / (B * B * bin_size * bin_size);
    i %= B * B * bin_size * bin_size;
    const int by = i / (B * bin_size * bin_size);
    i %= B * bin_size * bin_size;
    const int bx = i / (bin_size * bin_size);
    i %= bin_size * bin_size;

    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= S || xi >= S)
      continue;

    const float xf = PixToNdc(xi, S);
    const float yf = PixToNdc(yi, S);

    // This part looks like the naive rasterization kernel, except we use
    // bin_points to only look at a subset of points already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pix q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;
    for (int m = 0; m < M; ++m) {
      const int p = bin_points[n * B * B * M + by * B * M + bx * M + m];
      if (p < 0) {
        // bin_points uses -1 as a sentinal value
        continue;
      }
      CheckPixelInsidePoint(
          points, p, q_size, q_max_z, q_max_idx, q, radius2, xf, yf, K);
    }
    // Now we've looked at all the points for this bin, so we can write
    // output for the current pixel.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis as the camera coordinates
    // assume that +Y is pointing up and +X is pointing left.
    const int yidx = S - 1 - yi;
    const int xidx = S - 1 - xi;

    const int pix_idx = n * S * S * K + yidx * S * K + xidx * K;
    for (int k = 0; k < q_size; ++k) {
      point_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist2;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> RasterizePointsFineCuda(
    const torch::Tensor& points, // (P, 3)
    const torch::Tensor& bin_points,
    const int image_size,
    const float radius,
    const int bin_size,
    const int points_per_pixel) {
  const int N = bin_points.size(0);
  const int B = bin_points.size(1); // num_bins
  const int M = bin_points.size(3);
  const int S = image_size;
  const int K = points_per_pixel;
  if (K > kMaxPointsPerPixel) {
    AT_ERROR("Must have num_closest <= 8");
  }
  auto int_opts = points.options().dtype(torch::kInt32);
  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor point_idxs = torch::full({N, S, S, K}, -1, int_opts);
  torch::Tensor zbuf = torch::full({N, S, S, K}, -1, float_opts);
  torch::Tensor pix_dists = torch::full({N, S, S, K}, -1, float_opts);

  const size_t blocks = 1024;
  const size_t threads = 64;
  RasterizePointsFineCudaKernel<<<blocks, threads>>>(
      points.contiguous().data_ptr<float>(),
      bin_points.contiguous().data_ptr<int32_t>(),
      radius,
      bin_size,
      N,
      B,
      M,
      S,
      K,
      point_idxs.contiguous().data_ptr<int32_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>());

  return std::make_tuple(point_idxs, zbuf, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO(T55115174) Add more documentation for backward kernel.
__global__ void RasterizePointsBackwardCudaKernel(
    const float* points, // (P, 3)
    const int32_t* idxs, // (N, H, W, K)
    const int N,
    const int P,
    const int H,
    const int W,
    const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_dists, // (N, H, W, K)
    float* grad_points) { // (P, 3)
  // Parallelized over each of K points per pixel, for each pixel in images of
  // size H * W, for each image in the batch of size N.
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid; i < N * H * W * K; i += num_threads) {
    // const int n = i / (H * W * K); // batch index (not needed).
    const int yxk = i % (H * W * K);
    const int yi = yxk / (W * K);
    const int xk = yxk % (W * K);
    const int xi = xk / K;
    // k = xk % K (We don't actually need k, but this would be it.)
    // Reverse ordering of X and Y axes.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;

    const float xf = PixToNdc(xidx, W);
    const float yf = PixToNdc(yidx, H);

    const int p = idxs[i];
    if (p < 0)
      continue;
    const float grad_dist2 = grad_dists[i];
    const int p_ind = p * 3; // index into packed points tensor
    const float px = points[p_ind + 0];
    const float py = points[p_ind + 1];
    const float dx = px - xf;
    const float dy = py - yf;
    const float grad_px = 2.0f * grad_dist2 * dx;
    const float grad_py = 2.0f * grad_dist2 * dy;
    const float grad_pz = grad_zbuf[i];
    atomicAdd(grad_points + p_ind + 0, grad_px);
    atomicAdd(grad_points + p_ind + 1, grad_py);
    atomicAdd(grad_points + p_ind + 2, grad_pz);
  }
}

torch::Tensor RasterizePointsBackwardCuda(
    const torch::Tensor& points, // (N, P, 3)
    const torch::Tensor& idxs, // (N, H, W, K)
    const torch::Tensor& grad_zbuf, // (N, H, W, K)
    const torch::Tensor& grad_dists) { // (N, H, W, K)
  const int P = points.size(0);
  const int N = idxs.size(0);
  const int H = idxs.size(1);
  const int W = idxs.size(2);
  const int K = idxs.size(3);

  torch::Tensor grad_points = torch::zeros({P, 3}, points.options());
  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizePointsBackwardCudaKernel<<<blocks, threads>>>(
      points.contiguous().data_ptr<float>(),
      idxs.contiguous().data_ptr<int32_t>(),
      N,
      P,
      H,
      W,
      K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_points.contiguous().data_ptr<float>());

  return grad_points;
}
