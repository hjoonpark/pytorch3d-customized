#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void alphaCompositeCudaForwardKernel(
    // clang-format off
    torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> result,
    const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> features,
    const torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> alphas,
    const torch::PackedTensorAccessor64<int64_t, 4, torch::RestrictPtrTraits> points_idx) {
  // clang-format on
  const int64_t batch_size = result.size(0);
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  // Get the batch and index
  const int batch = blockIdx.x;

  const int num_pixels = C * W * H;
  const int num_threads = gridDim.y * blockDim.x;
  const int tid = blockIdx.y * blockDim.x + threadIdx.x;

  // Iterate over each feature in each pixel
  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    int ch = pid / (W * H);
    int j = (pid % (W * H)) / H;
    int i = (pid % (W * H)) % H;

    // alphacomposite the different values
    float cum_alpha = 1.;
    // Iterate through the closest K points for this pixel
    for (int k = 0; k < points_idx.size(1); ++k) {
      int n_idx = points_idx[batch][k][j][i];

      // Sentinel value is -1 indicating no point overlaps the pixel
      if (n_idx < 0) {
        continue;
      }

      float alpha = alphas[batch][k][j][i];
      // TODO(gkioxari) It might be more efficient to have threads write in a
      // local variable, and move atomicAdd outside of the loop such that
      // atomicAdd is executed once per thread.
      atomicAdd(
          &result[batch][ch][j][i], features[ch][n_idx] * cum_alpha * alpha);
      cum_alpha = cum_alpha * (1 - alpha);
    }
  }
}

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void alphaCompositeCudaBackwardKernel(
    // clang-format off
    torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> grad_features,
    torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> grad_alphas,
    const torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> grad_outputs,
    const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> features,
    const torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> alphas,
    const torch::PackedTensorAccessor64<int64_t, 4, torch::RestrictPtrTraits> points_idx) {
  // clang-format on
  const int64_t batch_size = points_idx.size(0);
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  // Get the batch and index
  const int batch = blockIdx.x;

  const int num_pixels = C * W * H;
  const int num_threads = gridDim.y * blockDim.x;
  const int tid = blockIdx.y * blockDim.x + threadIdx.x;

  // Parallelize over each feature in each pixel in images of size H * W,
  // for each image in the batch of size batch_size
  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    int ch = pid / (W * H);
    int j = (pid % (W * H)) / H;
    int i = (pid % (W * H)) % H;

    // alphacomposite the different values
    float cum_alpha = 1.;
    // Iterate through the closest K points for this pixel
    for (int k = 0; k < points_idx.size(1); ++k) {
      int n_idx = points_idx[batch][k][j][i];

      // Sentinel value is -1 indicating no point overlaps the pixel
      if (n_idx < 0) {
        continue;
      }
      float alpha = alphas[batch][k][j][i];

      // TODO(gkioxari) It might be more efficient to have threads write in a
      // local variable, and move atomicAdd outside of the loop such that
      // atomicAdd is executed once per thread.
      atomicAdd(
          &grad_alphas[batch][k][j][i],
          cum_alpha * features[ch][n_idx] * grad_outputs[batch][ch][j][i]);
      atomicAdd(
          &grad_features[ch][n_idx],
          cum_alpha * alpha * grad_outputs[batch][ch][j][i]);

      // Iterate over all (K-1) nearest points to update gradient
      for (int t = 0; t < k; ++t) {
        int t_idx = points_idx[batch][t][j][i];
        // Sentinel value is -1, indicating no point overlaps this pixel
        if (t_idx < 0) {
          continue;
        }
        float alpha_tvalue = alphas[batch][t][j][i];
        // TODO(gkioxari) It might be more efficient to have threads write in a
        // local variable, and move atomicAdd outside of the loop such that
        // atomicAdd is executed once per thread.
        atomicAdd(
            &grad_alphas[batch][t][j][i],
            -grad_outputs[batch][ch][j][i] * features[ch][n_idx] * cum_alpha *
                alpha / (1 - alpha_tvalue));
      }

      cum_alpha = cum_alpha * (1 - alphas[batch][k][j][i]);
    }
  }
}

torch::Tensor alphaCompositeCudaForward(
    const torch::Tensor& features,
    const torch::Tensor& alphas,
    const torch::Tensor& points_idx) {
  const int64_t batch_size = points_idx.size(0);
  const int64_t C = features.size(0);
  const int64_t H = points_idx.size(2);
  const int64_t W = points_idx.size(3);

  auto result = torch::zeros({batch_size, C, H, W}, features.options());

  const dim3 threadsPerBlock(64);
  const dim3 numBlocks(batch_size, 1024 / batch_size + 1);

  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  alphaCompositeCudaForwardKernel<<<numBlocks, threadsPerBlock>>>(
      // clang-format off
      result.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
      features.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
      alphas.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
      points_idx.packed_accessor64<int64_t, 4, torch::RestrictPtrTraits>());
  // clang-format on

  return result;
}

std::tuple<torch::Tensor, torch::Tensor> alphaCompositeCudaBackward(
    const torch::Tensor& grad_outputs,
    const torch::Tensor& features,
    const torch::Tensor& alphas,
    const torch::Tensor& points_idx) {
  auto grad_features = torch::zeros_like(features);
  auto grad_alphas = torch::zeros_like(alphas);

  const int64_t bs = alphas.size(0);

  const dim3 threadsPerBlock(64);
  const dim3 numBlocks(bs, 1024 / bs + 1);

  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  alphaCompositeCudaBackwardKernel<<<numBlocks, threadsPerBlock>>>(
      // clang-format off
      grad_features.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
      grad_alphas.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
      grad_outputs.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
      features.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
      alphas.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
      points_idx.packed_accessor64<int64_t, 4, torch::RestrictPtrTraits>());
  // clang-format on

  return std::make_tuple(grad_features, grad_alphas);
}
