#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <ATen/ATen.h>
#include <torch/extension.h>

// Kernel for inputs_packed of shape (F, D), where D > 1
template <typename scalar_t>
__global__ void PackedToPaddedKernel(
    const scalar_t* __restrict__ inputs_packed,
    const int64_t* __restrict__ first_idxs,
    scalar_t* __restrict__ inputs_padded,
    const size_t batch_size,
    const size_t max_size,
    const size_t num_inputs,
    const size_t D) {
  // Batch elements split evenly across blocks (num blocks = batch_size) and
  // values for each element split across threads in the block. Each thread adds
  // the values of its respective input elements to the global inputs_padded
  // tensor.
  const size_t tid = threadIdx.x;
  const size_t batch_idx = blockIdx.x;

  const int64_t start = first_idxs[batch_idx];
  const int64_t end =
      batch_idx + 1 < batch_size ? first_idxs[batch_idx + 1] : num_inputs;
  const int num = end - start;
  for (size_t f = tid; f < num; f += blockDim.x) {
    for (size_t j = 0; j < D; ++j) {
      inputs_padded[batch_idx * max_size * D + f * D + j] =
          inputs_packed[(start + f) * D + j];
    }
  }
}

// Kernel for inputs of shape (F, 1)
template <typename scalar_t>
__global__ void PackedToPaddedKernelD1(
    const scalar_t* __restrict__ inputs_packed,
    const int64_t* __restrict__ first_idxs,
    scalar_t* __restrict__ inputs_padded,
    const size_t batch_size,
    const size_t max_size,
    const size_t num_inputs) {
  // Batch elements split evenly across blocks (num blocks = batch_size) and
  // values for each element split across threads in the block. Each thread adds
  // the values of its respective input elements to the global inputs_padded
  // tensor.
  const size_t tid = threadIdx.x;
  const size_t batch_idx = blockIdx.x;

  const int64_t start = first_idxs[batch_idx];
  const int64_t end =
      batch_idx + 1 < batch_size ? first_idxs[batch_idx + 1] : num_inputs;
  const int num = end - start;
  for (size_t f = tid; f < num; f += blockDim.x) {
    inputs_padded[batch_idx * max_size + f] = inputs_packed[start + f];
  }
}

// Kernel for inputs_padded of shape (B, F, D), where D > 1
template <typename scalar_t>
__global__ void PaddedToPackedKernel(
    const scalar_t* __restrict__ inputs_padded,
    const int64_t* __restrict__ first_idxs,
    scalar_t* __restrict__ inputs_packed,
    const size_t batch_size,
    const size_t max_size,
    const size_t num_inputs,
    const size_t D) {
  // Batch elements split evenly across blocks (num blocks = batch_size) and
  // values for each element split across threads in the block. Each thread adds
  // the values of its respective input elements to the global inputs_packed
  // tensor.
  const size_t tid = threadIdx.x;
  const size_t batch_idx = blockIdx.x;

  const int64_t start = first_idxs[batch_idx];
  const int64_t end =
      batch_idx + 1 < batch_size ? first_idxs[batch_idx + 1] : num_inputs;
  const int num = end - start;
  for (size_t f = tid; f < num; f += blockDim.x) {
    for (size_t j = 0; j < D; ++j) {
      inputs_packed[(start + f) * D + j] =
          inputs_padded[batch_idx * max_size * D + f * D + j];
    }
  }
}

// Kernel for inputs_padded of shape (B, F, 1)
template <typename scalar_t>
__global__ void PaddedToPackedKernelD1(
    const scalar_t* __restrict__ inputs_padded,
    const int64_t* __restrict__ first_idxs,
    scalar_t* __restrict__ inputs_packed,
    const size_t batch_size,
    const size_t max_size,
    const size_t num_inputs) {
  // Batch elements split evenly across blocks (num blocks = batch_size) and
  // values for each element split across threads in the block. Each thread adds
  // the values of its respective input elements to the global inputs_packed
  // tensor.
  const size_t tid = threadIdx.x;
  const size_t batch_idx = blockIdx.x;

  const int64_t start = first_idxs[batch_idx];
  const int64_t end =
      batch_idx + 1 < batch_size ? first_idxs[batch_idx + 1] : num_inputs;
  const int num = end - start;
  for (size_t f = tid; f < num; f += blockDim.x) {
    inputs_packed[start + f] = inputs_padded[batch_idx * max_size + f];
  }
}

at::Tensor PackedToPaddedCuda(
    const at::Tensor inputs_packed,
    const at::Tensor first_idxs,
    const int64_t max_size) {
  const int64_t num_inputs = inputs_packed.size(0);
  const int64_t batch_size = first_idxs.size(0);

  AT_ASSERTM(
      inputs_packed.dim() == 2, "inputs_packed must be a 2-dimensional tensor");
  const int64_t D = inputs_packed.size(1);
  at::Tensor inputs_padded =
      at::zeros({batch_size, max_size, D}, inputs_packed.options());

  const int threads = 512;
  const int blocks = batch_size;
  if (D == 1) {
    AT_DISPATCH_FLOATING_TYPES(
        inputs_packed.scalar_type(), "packed_to_padded_d1_kernel", ([&] {
          PackedToPaddedKernelD1<scalar_t><<<blocks, threads>>>(
              inputs_packed.data_ptr<scalar_t>(),
              first_idxs.data_ptr<int64_t>(),
              inputs_padded.data_ptr<scalar_t>(),
              batch_size,
              max_size,
              num_inputs);
        }));
  } else {
    AT_DISPATCH_FLOATING_TYPES(
        inputs_packed.scalar_type(), "packed_to_padded_kernel", ([&] {
          PackedToPaddedKernel<scalar_t><<<blocks, threads>>>(
              inputs_packed.data_ptr<scalar_t>(),
              first_idxs.data_ptr<int64_t>(),
              inputs_padded.data_ptr<scalar_t>(),
              batch_size,
              max_size,
              num_inputs,
              D);
        }));
  }

  return inputs_padded;
}

at::Tensor PaddedToPackedCuda(
    const at::Tensor inputs_padded,
    const at::Tensor first_idxs,
    const int64_t num_inputs) {
  const int64_t batch_size = inputs_padded.size(0);
  const int64_t max_size = inputs_padded.size(1);

  AT_ASSERTM(batch_size == first_idxs.size(0), "sizes mismatch");
  AT_ASSERTM(
      inputs_padded.dim() == 3,
      "inputs_padded  must be a 3-dimensional tensor");
  const int64_t D = inputs_padded.size(2);

  at::Tensor inputs_packed =
      at::zeros({num_inputs, D}, inputs_padded.options());

  const int threads = 512;
  const int blocks = batch_size;

  if (D == 1) {
    AT_DISPATCH_FLOATING_TYPES(
        inputs_padded.scalar_type(), "padded_to_packed_d1_kernel", ([&] {
          PaddedToPackedKernelD1<scalar_t><<<blocks, threads>>>(
              inputs_padded.data_ptr<scalar_t>(),
              first_idxs.data_ptr<int64_t>(),
              inputs_packed.data_ptr<scalar_t>(),
              batch_size,
              max_size,
              num_inputs);
        }));
  } else {
    AT_DISPATCH_FLOATING_TYPES(
        inputs_padded.scalar_type(), "padded_to_packed_kernel", ([&] {
          PaddedToPackedKernel<scalar_t><<<blocks, threads>>>(
              inputs_padded.data_ptr<scalar_t>(),
              first_idxs.data_ptr<int64_t>(),
              inputs_packed.data_ptr<scalar_t>(),
              batch_size,
              max_size,
              num_inputs,
              D);
        }));
  }

  return inputs_packed;
}
