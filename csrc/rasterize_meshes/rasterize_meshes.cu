#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <float.h>
#include <math.h>
#include <thrust/tuple.h>
#include <torch/extension.h>
#include <cstdio>
#include <tuple>
#include "rasterize_points/bitmask.cuh"
#include "rasterize_points/rasterization_utils.cuh"
#include "utils/float_math.cuh"
#include "utils/geometry_utils.cuh"

namespace {
// A structure for holding details about a pixel.
struct Pixel {
  float z;
  int64_t idx;
  float dist;
  float3 bary;
};

__device__ bool operator<(const Pixel& a, const Pixel& b) {
  return a.z < b.z;
}

__device__ float FloatMin3(const float p1, const float p2, const float p3) {
  return fminf(p1, fminf(p2, p3));
}

__device__ float FloatMax3(const float p1, const float p2, const float p3) {
  return fmaxf(p1, fmaxf(p2, p3));
}

// Get the xyz coordinates of the three vertices for the face given by the
// index face_idx into face_verts.
__device__ thrust::tuple<float3, float3, float3> GetSingleFaceVerts(
    const float* face_verts,
    int face_idx) {
  const float x0 = face_verts[face_idx * 9 + 0];
  const float y0 = face_verts[face_idx * 9 + 1];
  const float z0 = face_verts[face_idx * 9 + 2];
  const float x1 = face_verts[face_idx * 9 + 3];
  const float y1 = face_verts[face_idx * 9 + 4];
  const float z1 = face_verts[face_idx * 9 + 5];
  const float x2 = face_verts[face_idx * 9 + 6];
  const float y2 = face_verts[face_idx * 9 + 7];
  const float z2 = face_verts[face_idx * 9 + 8];

  const float3 v0xyz = make_float3(x0, y0, z0);
  const float3 v1xyz = make_float3(x1, y1, z1);
  const float3 v2xyz = make_float3(x2, y2, z2);

  return thrust::make_tuple(v0xyz, v1xyz, v2xyz);
}

// Get the min/max x/y/z values for the face given by vertices v0, v1, v2.
__device__ thrust::tuple<float2, float2, float2>
GetFaceBoundingBox(float3 v0, float3 v1, float3 v2) {
  const float xmin = FloatMin3(v0.x, v1.x, v2.x);
  const float ymin = FloatMin3(v0.y, v1.y, v2.y);
  const float zmin = FloatMin3(v0.z, v1.z, v2.z);
  const float xmax = FloatMax3(v0.x, v1.x, v2.x);
  const float ymax = FloatMax3(v0.y, v1.y, v2.y);
  const float zmax = FloatMax3(v0.z, v1.z, v2.z);

  return thrust::make_tuple(
      make_float2(xmin, xmax),
      make_float2(ymin, ymax),
      make_float2(zmin, zmax));
}

// Check if the point (px, py) lies outside the face bounding box face_bbox.
// Return true if the point is outside.
__device__ bool CheckPointOutsideBoundingBox(
    float3 v0,
    float3 v1,
    float3 v2,
    float blur_radius,
    float2 pxy) {
  const auto bbox = GetFaceBoundingBox(v0, v1, v2);
  const float2 xlims = thrust::get<0>(bbox);
  const float2 ylims = thrust::get<1>(bbox);
  const float2 zlims = thrust::get<2>(bbox);

  const float x_min = xlims.x - blur_radius;
  const float y_min = ylims.x - blur_radius;
  const float x_max = xlims.y + blur_radius;
  const float y_max = ylims.y + blur_radius;

  // Check if the current point is oustside the triangle bounding box.
  return (pxy.x > x_max || pxy.x < x_min || pxy.y > y_max || pxy.y < y_min);
}

// This function checks if a pixel given by xy location pxy lies within the
// face with index face_idx in face_verts. One of the inputs is a list (q)
// which contains Pixel structs with the indices of the faces which intersect
// with this pixel sorted by closest z distance. If the point pxy lies in the
// face, the list (q) is updated and re-orderered in place. In addition
// the auxillary variables q_size, q_max_z and q_max_idx are also modified.
// This code is shared between RasterizeMeshesNaiveCudaKernel and
// RasterizeMeshesFineCudaKernel.
template <typename FaceQ>
__device__ void CheckPixelInsideFace(
    const float* face_verts, // (F, 3, 3)
    const int face_idx,
    int& q_size,
    float& q_max_z,
    int& q_max_idx,
    FaceQ& q,
    const float blur_radius,
    const float2 pxy, // Coordinates of the pixel
    const int K,
    const bool perspective_correct) {
  const auto v012 = GetSingleFaceVerts(face_verts, face_idx);
  const float3 v0 = thrust::get<0>(v012);
  const float3 v1 = thrust::get<1>(v012);
  const float3 v2 = thrust::get<2>(v012);

  // Only need xy for barycentric coordinates and distance calculations.
  const float2 v0xy = make_float2(v0.x, v0.y);
  const float2 v1xy = make_float2(v1.x, v1.y);
  const float2 v2xy = make_float2(v2.x, v2.y);

  // Perform checks and skip if:
  // 1. the face is behind the camera
  // 2. the face has very small face area
  // 3. the pixel is outside the face bbox
  const float zmax = FloatMax3(v0.z, v1.z, v2.z);
  const bool outside_bbox = CheckPointOutsideBoundingBox(
      v0, v1, v2, sqrt(blur_radius), pxy); // use sqrt of blur for bbox
  const float face_area = EdgeFunctionForward(v0xy, v1xy, v2xy);
  const bool zero_face_area =
      (face_area <= kEpsilon && face_area >= -1.0f * kEpsilon);

  if (zmax < 0 || outside_bbox || zero_face_area) {
    return;
  }

  // Calculate barycentric coords and euclidean dist to triangle.
  const float3 p_bary0 = BarycentricCoordsForward(pxy, v0xy, v1xy, v2xy);
  const float3 p_bary = !perspective_correct
      ? p_bary0
      : BarycentricPerspectiveCorrectionForward(p_bary0, v0.z, v1.z, v2.z);

  const float pz = p_bary.x * v0.z + p_bary.y * v1.z + p_bary.z * v2.z;
  if (pz < 0) {
    return; // Face is behind the image plane.
  }

  // Get abs squared distance
  const float dist = PointTriangleDistanceForward(pxy, v0xy, v1xy, v2xy);

  // Use the bary coordinates to determine if the point is inside the face.
  const bool inside = p_bary.x > 0.0f && p_bary.y > 0.0f && p_bary.z > 0.0f;
  const float signed_dist = inside ? -dist : dist;

  // Check if pixel is outside blur region
  if (!inside && dist >= blur_radius) {
    return;
  }

  if (q_size < K) {
    // Just insert it.
    q[q_size] = {pz, face_idx, signed_dist, p_bary};
    if (pz > q_max_z) {
      q_max_z = pz;
      q_max_idx = q_size;
    }
    q_size++;
  } else if (pz < q_max_z) {
    // Overwrite the old max, and find the new max.
    q[q_max_idx] = {pz, face_idx, signed_dist, p_bary};
    q_max_z = pz;
    for (int i = 0; i < K; i++) {
      if (q[i].z > q_max_z) {
        q_max_z = q[i].z;
        q_max_idx = i;
      }
    }
  }
}
} // namespace

// ****************************************************************************
// *                          NAIVE RASTERIZATION                      *
// ****************************************************************************
__global__ void RasterizeMeshesNaiveCudaKernel(
    const float* face_verts,
    const int64_t* mesh_to_face_first_idx,
    const int64_t* num_faces_per_mesh,
    const float blur_radius,
    const bool perspective_correct,
    const int N,
    const int H,
    const int W,
    const int K,
    int64_t* face_idxs,
    float* zbuf,
    float* pix_dists,
    float* bary) {
  // Simple version: One thread per output pixel
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = tid; i < N * H * W; i += num_threads) {
    // Convert linear index to 3D index
    const int n = i / (H * W); // batch index.
    const int pix_idx = i % (H * W);

    // Reverse ordering of X and Y axes
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    // screen coordinates to ndc coordiantes of pixel.
    const float xf = PixToNdc(xi, W);
    const float yf = PixToNdc(yi, H);
    const float2 pxy = make_float2(xf, yf);

    // For keeping track of the K closest points we want a data structure
    // that (1) gives O(1) access to the closest point for easy comparisons,
    // and (2) allows insertion of new elements. In the CPU version we use
    // std::priority_queue; then (2) is O(log K). We can't use STL
    // containers in CUDA; we could roll our own max heap in an array, but
    // that would likely have a lot of warp divergence so we do something
    // simpler instead: keep the elements in an unsorted array, but keep
    // track of the max value and the index of the max value. Then (1) is
    // still O(1) time, while (2) is O(K) with a clean loop. Since K <= 8
    // this should be fast enough for our purposes.
    Pixel q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;

    // Using the batch index of the thread get the start and stop
    // indices for the faces.
    const int64_t face_start_idx = mesh_to_face_first_idx[n];
    const int64_t face_stop_idx = face_start_idx + num_faces_per_mesh[n];

    // Loop through the faces in the mesh.
    for (int f = face_start_idx; f < face_stop_idx; ++f) {
      // Check if the pixel pxy is inside the face bounding box and if it is,
      // update q, q_size, q_max_z and q_max_idx in place.
      CheckPixelInsideFace(
          face_verts,
          f,
          q_size,
          q_max_z,
          q_max_idx,
          q,
          blur_radius,
          pxy,
          K,
          perspective_correct);
    }

    // TODO: make sorting an option as only top k is needed, not sorted values.
    BubbleSort(q, q_size);
    int idx = n * H * W * K + pix_idx * K;
    for (int k = 0; k < q_size; ++k) {
      face_idxs[idx + k] = q[k].idx;
      zbuf[idx + k] = q[k].z;
      pix_dists[idx + k] = q[k].dist;
      bary[(idx + k) * 3 + 0] = q[k].bary.x;
      bary[(idx + k) * 3 + 1] = q[k].bary.y;
      bary[(idx + k) * 3 + 2] = q[k].bary.z;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeMeshesNaiveCuda(
    const torch::Tensor& face_verts,
    const torch::Tensor& mesh_to_faces_packed_first_idx,
    const torch::Tensor& num_faces_per_mesh,
    const int image_size,
    const float blur_radius,
    const int num_closest,
    const bool perspective_correct) {
  if (face_verts.ndimension() != 3 || face_verts.size(1) != 3 ||
      face_verts.size(2) != 3) {
    AT_ERROR("face_verts must have dimensions (num_faces, 3, 3)");
  }
  if (num_faces_per_mesh.size(0) != mesh_to_faces_packed_first_idx.size(0)) {
    AT_ERROR(
        "num_faces_per_mesh must have save size first dimension as mesh_to_faces_packed_first_idx");
  }

  if (num_closest > kMaxPointsPerPixel) {
    std::stringstream ss;
    ss << "Must have points_per_pixel <= " << kMaxPointsPerPixel;
    AT_ERROR(ss.str());
  }

  const int N = num_faces_per_mesh.size(0); // batch size.
  const int H = image_size; // Assume square images.
  const int W = image_size;
  const int K = num_closest;

  auto long_opts = face_verts.options().dtype(torch::kInt64);
  auto float_opts = face_verts.options().dtype(torch::kFloat32);

  torch::Tensor face_idxs = torch::full({N, H, W, K}, -1, long_opts);
  torch::Tensor zbuf = torch::full({N, H, W, K}, -1, float_opts);
  torch::Tensor pix_dists = torch::full({N, H, W, K}, -1, float_opts);
  torch::Tensor bary = torch::full({N, H, W, K, 3}, -1, float_opts);

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesNaiveCudaKernel<<<blocks, threads>>>(
      face_verts.contiguous().data_ptr<float>(),
      mesh_to_faces_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_faces_per_mesh.contiguous().data_ptr<int64_t>(),
      blur_radius,
      perspective_correct,
      N,
      H,
      W,
      K,
      face_idxs.contiguous().data_ptr<int64_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>(),
      bary.contiguous().data_ptr<float>());

  return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
}

// ****************************************************************************
// *                            BACKWARD PASS                                 *
// ****************************************************************************
// TODO: benchmark parallelizing over faces_verts instead of over pixels.
__global__ void RasterizeMeshesBackwardCudaKernel(
    const float* face_verts, // (F, 3, 3)
    const int64_t* pix_to_face, // (N, H, W, K)
    const bool perspective_correct,
    const int N,
    const int H,
    const int W,
    const int K,
    const float* grad_zbuf, // (N, H, W, K)
    const float* grad_bary, // (N, H, W, K, 3)
    const float* grad_dists, // (N, H, W, K)
    float* grad_face_verts) { // (F, 3, 3)

  // Parallelize over each pixel in images of
  // size H * W, for each image in the batch of size N.
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int t_i = tid; t_i < N * H * W; t_i += num_threads) {
    // Convert linear index to 3D index
    const int n = t_i / (H * W); // batch index.
    const int pix_idx = t_i % (H * W);

    // Reverse ordering of X and Y axes.
    const int yi = H - 1 - pix_idx / W;
    const int xi = W - 1 - pix_idx % W;

    const float xf = PixToNdc(xi, W);
    const float yf = PixToNdc(yi, H);
    const float2 pxy = make_float2(xf, yf);

    // Loop over all the faces for this pixel.
    for (int k = 0; k < K; k++) {
      // Index into (N, H, W, K, :) grad tensors
      // pixel index + top k index
      int i = n * H * W * K + pix_idx * K + k;

      const int f = pix_to_face[i];
      if (f < 0) {
        continue; // padded face.
      }
      // Get xyz coordinates of the three face vertices.
      const auto v012 = GetSingleFaceVerts(face_verts, f);
      const float3 v0 = thrust::get<0>(v012);
      const float3 v1 = thrust::get<1>(v012);
      const float3 v2 = thrust::get<2>(v012);

      // Only neex xy for barycentric coordinate and distance calculations.
      const float2 v0xy = make_float2(v0.x, v0.y);
      const float2 v1xy = make_float2(v1.x, v1.y);
      const float2 v2xy = make_float2(v2.x, v2.y);

      // Get upstream gradients for the face.
      const float grad_dist_upstream = grad_dists[i];
      const float grad_zbuf_upstream = grad_zbuf[i];
      const float grad_bary_upstream_w0 = grad_bary[i * 3 + 0];
      const float grad_bary_upstream_w1 = grad_bary[i * 3 + 1];
      const float grad_bary_upstream_w2 = grad_bary[i * 3 + 2];
      const float3 grad_bary_upstream = make_float3(
          grad_bary_upstream_w0, grad_bary_upstream_w1, grad_bary_upstream_w2);

      const float3 bary0 = BarycentricCoordsForward(pxy, v0xy, v1xy, v2xy);
      const float3 bary = !perspective_correct
          ? bary0
          : BarycentricPerspectiveCorrectionForward(bary0, v0.z, v1.z, v2.z);
      const bool inside = bary.x > 0.0f && bary.y > 0.0f && bary.z > 0.0f;
      const float sign = inside ? -1.0f : 1.0f;

      // TODO(T52813608) Add support for non-square images.
      auto grad_dist_f = PointTriangleDistanceBackward(
          pxy, v0xy, v1xy, v2xy, sign * grad_dist_upstream);
      const float2 ddist_d_v0 = thrust::get<1>(grad_dist_f);
      const float2 ddist_d_v1 = thrust::get<2>(grad_dist_f);
      const float2 ddist_d_v2 = thrust::get<3>(grad_dist_f);

      // Upstream gradient for barycentric coords from zbuf calculation:
      // zbuf = bary_w0 * z0 + bary_w1 * z1 + bary_w2 * z2
      // Therefore
      // d_zbuf/d_bary_w0 = z0
      // d_zbuf/d_bary_w1 = z1
      // d_zbuf/d_bary_w2 = z2
      const float3 d_zbuf_d_bary = make_float3(v0.z, v1.z, v2.z);

      // Total upstream barycentric gradients are the sum of
      // external upstream gradients and contribution from zbuf.
      const float3 grad_bary_f_sum =
          (grad_bary_upstream + grad_zbuf_upstream * d_zbuf_d_bary);
      float3 grad_bary0 = grad_bary_f_sum;
      float dz0_persp = 0.0f, dz1_persp = 0.0f, dz2_persp = 0.0f;
      if (perspective_correct) {
        auto perspective_grads = BarycentricPerspectiveCorrectionBackward(
            bary0, v0.z, v1.z, v2.z, grad_bary_f_sum);
        grad_bary0 = thrust::get<0>(perspective_grads);
        dz0_persp = thrust::get<1>(perspective_grads);
        dz1_persp = thrust::get<2>(perspective_grads);
        dz2_persp = thrust::get<3>(perspective_grads);
      }
      auto grad_bary_f =
          BarycentricCoordsBackward(pxy, v0xy, v1xy, v2xy, grad_bary0);
      const float2 dbary_d_v0 = thrust::get<1>(grad_bary_f);
      const float2 dbary_d_v1 = thrust::get<2>(grad_bary_f);
      const float2 dbary_d_v2 = thrust::get<3>(grad_bary_f);

      atomicAdd(grad_face_verts + f * 9 + 0, dbary_d_v0.x + ddist_d_v0.x);
      atomicAdd(grad_face_verts + f * 9 + 1, dbary_d_v0.y + ddist_d_v0.y);
      atomicAdd(
          grad_face_verts + f * 9 + 2, grad_zbuf_upstream * bary.x + dz0_persp);
      atomicAdd(grad_face_verts + f * 9 + 3, dbary_d_v1.x + ddist_d_v1.x);
      atomicAdd(grad_face_verts + f * 9 + 4, dbary_d_v1.y + ddist_d_v1.y);
      atomicAdd(
          grad_face_verts + f * 9 + 5, grad_zbuf_upstream * bary.y + dz1_persp);
      atomicAdd(grad_face_verts + f * 9 + 6, dbary_d_v2.x + ddist_d_v2.x);
      atomicAdd(grad_face_verts + f * 9 + 7, dbary_d_v2.y + ddist_d_v2.y);
      atomicAdd(
          grad_face_verts + f * 9 + 8, grad_zbuf_upstream * bary.z + dz2_persp);
    }
  }
}

torch::Tensor RasterizeMeshesBackwardCuda(
    const torch::Tensor& face_verts, // (F, 3, 3)
    const torch::Tensor& pix_to_face, // (N, H, W, K)
    const torch::Tensor& grad_zbuf, // (N, H, W, K)
    const torch::Tensor& grad_bary, // (N, H, W, K, 3)
    const torch::Tensor& grad_dists, // (N, H, W, K)
    const bool perspective_correct) {
  const int F = face_verts.size(0);
  const int N = pix_to_face.size(0);
  const int H = pix_to_face.size(1);
  const int W = pix_to_face.size(2);
  const int K = pix_to_face.size(3);

  torch::Tensor grad_face_verts = torch::zeros({F, 3, 3}, face_verts.options());
  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesBackwardCudaKernel<<<blocks, threads>>>(
      face_verts.contiguous().data_ptr<float>(),
      pix_to_face.contiguous().data_ptr<int64_t>(),
      perspective_correct,
      N,
      H,
      W,
      K,
      grad_zbuf.contiguous().data_ptr<float>(),
      grad_bary.contiguous().data_ptr<float>(),
      grad_dists.contiguous().data_ptr<float>(),
      grad_face_verts.contiguous().data_ptr<float>());

  return grad_face_verts;
}

// ****************************************************************************
// *                          COARSE RASTERIZATION                            *
// ****************************************************************************

__global__ void RasterizeMeshesCoarseCudaKernel(
    const float* face_verts,
    const int64_t* mesh_to_face_first_idx,
    const int64_t* num_faces_per_mesh,
    const float blur_radius,
    const int N,
    const int F,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_faces_per_bin,
    int* faces_per_bin,
    int* bin_faces) {
  extern __shared__ char sbuf[];
  const int M = max_faces_per_bin;
  const int num_bins = 1 + (W - 1) / bin_size; // Integer divide round up
  const float half_pix = 1.0f / W; // Size of half a pixel in NDC units
  // This is a boolean array of shape (num_bins, num_bins, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins, num_bins, chunk_size);

  // Have each block handle a chunk of faces
  const int chunks_per_batch = 1 + (F - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;

  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch; // batch index
    const int chunk_idx = chunk % chunks_per_batch;
    const int face_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();
    const int64_t mesh_face_start_idx = mesh_to_face_first_idx[batch_idx];
    const int64_t mesh_face_stop_idx =
        mesh_face_start_idx + num_faces_per_mesh[batch_idx];

    // Have each thread handle a different face within the chunk
    for (int f = threadIdx.x; f < chunk_size; f += blockDim.x) {
      const int f_idx = face_start_idx + f;

      // Check if face index corresponds to the mesh in the batch given by
      // batch_idx
      if (f_idx >= mesh_face_stop_idx || f_idx < mesh_face_start_idx) {
        continue;
      }

      // Get xyz coordinates of the three face vertices.
      const auto v012 = GetSingleFaceVerts(face_verts, f_idx);
      const float3 v0 = thrust::get<0>(v012);
      const float3 v1 = thrust::get<1>(v012);
      const float3 v2 = thrust::get<2>(v012);

      // Compute screen-space bbox for the triangle expanded by blur.
      float xmin = FloatMin3(v0.x, v1.x, v2.x) - sqrt(blur_radius);
      float ymin = FloatMin3(v0.y, v1.y, v2.y) - sqrt(blur_radius);
      float xmax = FloatMax3(v0.x, v1.x, v2.x) + sqrt(blur_radius);
      float ymax = FloatMax3(v0.y, v1.y, v2.y) + sqrt(blur_radius);
      float zmax = FloatMax3(v0.z, v1.z, v2.z);

      if (zmax < 0) {
        continue; // Face is behind the camera.
      }

      // Brute-force search over all bins; TODO(T54294966) something smarter.
      for (int by = 0; by < num_bins; ++by) {
        // Y coordinate of the top and bottom of the bin.
        // PixToNdc gives the location of the center of each pixel, so we
        // need to add/subtract a half pixel to get the true extent of the bin.
        // Reverse ordering of Y axis so that +Y is upwards in the image.
        const float bin_y_min = PixToNdc(by * bin_size, H) - half_pix;
        const float bin_y_max = PixToNdc((by + 1) * bin_size - 1, H) + half_pix;
        const bool y_overlap = (ymin <= bin_y_max) && (bin_y_min < ymax);

        for (int bx = 0; bx < num_bins; ++bx) {
          // X coordinate of the left and right of the bin.
          // Reverse ordering of x axis so that +X is left.
          const float bin_x_max =
              PixToNdc((bx + 1) * bin_size - 1, W) + half_pix;
          const float bin_x_min = PixToNdc(bx * bin_size, W) - half_pix;

          const bool x_overlap = (xmin <= bin_x_max) && (bin_x_min < xmax);
          if (y_overlap && x_overlap) {
            binmask.set(by, bx, f);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every face in the current chunk. We need to
    // count the number of faces in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins * num_bins; byx += blockDim.x) {
      const int by = byx / num_bins;
      const int bx = byx % num_bins;
      const int count = binmask.count(by, bx);
      const int faces_per_bin_idx =
          batch_idx * num_bins * num_bins + by * num_bins + bx;

      // This atomically increments the (global) number of faces found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_faces array for the
      // faces in the current chunk that fall into this bin.
      const int start = atomicAdd(faces_per_bin + faces_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_faces.
      int next_idx = batch_idx * num_bins * num_bins * M + by * num_bins * M +
          bx * M + start;
      for (int f = 0; f < chunk_size; ++f) {
        if (binmask.get(by, bx, f)) {
          // TODO(T54296346) find the correct method for handling errors in
          // CUDA. Throw an error if num_faces_per_bin > max_faces_per_bin.
          // Either decrease bin size or increase max_faces_per_bin
          bin_faces[next_idx] = face_start_idx + f;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

torch::Tensor RasterizeMeshesCoarseCuda(
    const torch::Tensor& face_verts,
    const torch::Tensor& mesh_to_face_first_idx,
    const torch::Tensor& num_faces_per_mesh,
    const int image_size,
    const float blur_radius,
    const int bin_size,
    const int max_faces_per_bin) {
  if (face_verts.ndimension() != 3 || face_verts.size(1) != 3 ||
      face_verts.size(2) != 3) {
    AT_ERROR("face_verts must have dimensions (num_faces, 3, 3)");
  }
  const int W = image_size;
  const int H = image_size;
  const int F = face_verts.size(0);
  const int N = num_faces_per_mesh.size(0);
  const int num_bins = 1 + (image_size - 1) / bin_size; // Divide round up.
  const int M = max_faces_per_bin;

  if (num_bins >= 22) {
    std::stringstream ss;
    ss << "Got " << num_bins << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = face_verts.options().dtype(torch::kInt32);
  torch::Tensor faces_per_bin = torch::zeros({N, num_bins, num_bins}, opts);
  torch::Tensor bin_faces = torch::full({N, num_bins, num_bins, M}, -1, opts);
  const int chunk_size = 512;
  const size_t shared_size = num_bins * num_bins * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizeMeshesCoarseCudaKernel<<<blocks, threads, shared_size>>>(
      face_verts.contiguous().data_ptr<float>(),
      mesh_to_face_first_idx.contiguous().data_ptr<int64_t>(),
      num_faces_per_mesh.contiguous().data_ptr<int64_t>(),
      blur_radius,
      N,
      F,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      faces_per_bin.contiguous().data_ptr<int32_t>(),
      bin_faces.contiguous().data_ptr<int32_t>());
  return bin_faces;
}

// ****************************************************************************
// *                            FINE RASTERIZATION                            *
// ****************************************************************************
__global__ void RasterizeMeshesFineCudaKernel(
    const float* face_verts, // (F, 3, 3)
    const int32_t* bin_faces, // (N, B, B, T)
    const float blur_radius,
    const int bin_size,
    const bool perspective_correct,
    const int N,
    const int B,
    const int M,
    const int H,
    const int W,
    const int K,
    int64_t* face_idxs, // (N, S, S, K)
    float* zbuf, // (N, S, S, K)
    float* pix_dists, // (N, S, S, K)
    float* bary // (N, S, S, K, 3)
) {
  // This can be more than S^2 if S % bin_size != 0
  int num_pixels = N * B * B * bin_size * bin_size;
  int num_threads = gridDim.x * blockDim.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int pid = tid; pid < num_pixels; pid += num_threads) {
    // Convert linear index into bin and pixel indices. We make the within
    // block pixel ids move the fastest, so that adjacent threads will fall
    // into the same bin; this should give them coalesced memory reads when
    // they read from faces and bin_faces.
    int i = pid;
    const int n = i / (B * B * bin_size * bin_size);
    i %= B * B * bin_size * bin_size;
    const int by = i / (B * bin_size * bin_size);
    i %= B * bin_size * bin_size;
    const int bx = i / (bin_size * bin_size);
    i %= bin_size * bin_size;
    const int yi = i / bin_size + by * bin_size;
    const int xi = i % bin_size + bx * bin_size;

    if (yi >= H || xi >= W)
      continue;

    const float xf = PixToNdc(xi, W);
    const float yf = PixToNdc(yi, H);
    const float2 pxy = make_float2(xf, yf);

    // This part looks like the naive rasterization kernel, except we use
    // bin_faces to only look at a subset of faces already known to fall
    // in this bin. TODO abstract out this logic into some data structure
    // that is shared by both kernels?
    Pixel q[kMaxPointsPerPixel];
    int q_size = 0;
    float q_max_z = -1000;
    int q_max_idx = -1;
    for (int m = 0; m < M; m++) {
      const int f = bin_faces[n * B * B * M + by * B * M + bx * M + m];
      if (f < 0) {
        continue; // bin_faces uses -1 as a sentinal value.
      }
      // Check if the pixel pxy is inside the face bounding box and if it is,
      // update q, q_size, q_max_z and q_max_idx in place.
      CheckPixelInsideFace(
          face_verts,
          f,
          q_size,
          q_max_z,
          q_max_idx,
          q,
          blur_radius,
          pxy,
          K,
          perspective_correct);
    }

    // Now we've looked at all the faces for this bin, so we can write
    // output for the current pixel.
    // TODO: make sorting an option as only top k is needed, not sorted values.
    BubbleSort(q, q_size);

    // Reverse ordering of the X and Y axis so that
    // in the image +Y is pointing up and +X is pointing left.
    const int yidx = H - 1 - yi;
    const int xidx = W - 1 - xi;
    const int pix_idx = n * H * W * K + yidx * H * K + xidx * K;
    for (int k = 0; k < q_size; k++) {
      face_idxs[pix_idx + k] = q[k].idx;
      zbuf[pix_idx + k] = q[k].z;
      pix_dists[pix_idx + k] = q[k].dist;
      bary[(pix_idx + k) * 3 + 0] = q[k].bary.x;
      bary[(pix_idx + k) * 3 + 1] = q[k].bary.y;
      bary[(pix_idx + k) * 3 + 2] = q[k].bary.z;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeMeshesFineCuda(
    const torch::Tensor& face_verts,
    const torch::Tensor& bin_faces,
    const int image_size,
    const float blur_radius,
    const int bin_size,
    const int faces_per_pixel,
    const bool perspective_correct) {
  if (face_verts.ndimension() != 3 || face_verts.size(1) != 3 ||
      face_verts.size(2) != 3) {
    AT_ERROR("face_verts must have dimensions (num_faces, 3, 3)");
  }
  if (bin_faces.ndimension() != 4) {
    AT_ERROR("bin_faces must have 4 dimensions");
  }
  const int N = bin_faces.size(0);
  const int B = bin_faces.size(1);
  const int M = bin_faces.size(3);
  const int K = faces_per_pixel;
  const int H = image_size; // Assume square images only.
  const int W = image_size;

  if (K > kMaxPointsPerPixel) {
    AT_ERROR("Must have num_closest <= 8");
  }
  auto long_opts = face_verts.options().dtype(torch::kInt64);
  auto float_opts = face_verts.options().dtype(torch::kFloat32);

  torch::Tensor face_idxs = torch::full({N, H, W, K}, -1, long_opts);
  torch::Tensor zbuf = torch::full({N, H, W, K}, -1, float_opts);
  torch::Tensor pix_dists = torch::full({N, H, W, K}, -1, float_opts);
  torch::Tensor bary = torch::full({N, H, W, K, 3}, -1, float_opts);

  const size_t blocks = 1024;
  const size_t threads = 64;

  RasterizeMeshesFineCudaKernel<<<blocks, threads>>>(
      face_verts.contiguous().data_ptr<float>(),
      bin_faces.contiguous().data_ptr<int32_t>(),
      blur_radius,
      bin_size,
      perspective_correct,
      N,
      B,
      M,
      H,
      W,
      K,
      face_idxs.contiguous().data_ptr<int64_t>(),
      zbuf.contiguous().data_ptr<float>(),
      pix_dists.contiguous().data_ptr<float>(),
      bary.contiguous().data_ptr<float>());

  return std::make_tuple(face_idxs, zbuf, bary, pix_dists);
}
